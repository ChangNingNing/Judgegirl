#include "hip/hip_runtime.h"
#include "labeling.h"
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

#define ThreadSize 256
#define LOGT 8
#define SeqSize 4

__global__ void myCudaCount(const char *text, int *pos, int n){
	int bid = blockIdx.x;
	int left = (blockIdx.y == 1)? bid * blockDim.x + ThreadSize/2: bid * blockDim.x;
	int tid = threadIdx.x;
	int id = left + tid;

	__shared__ int BIT[ThreadSize][LOGT];
	__shared__ int iBIT[ThreadSize];
	int seqIndex = id * SeqSize;
	int seq[SeqSize];

	if (seqIndex < n){
		// Transform
		int count = 1;
		for (int i=0; i<SeqSize; i++, count++){
			if (seqIndex + i < n){
				if (text[seqIndex + i] <= ' ')
					count = 0;
			}
			else{
				count = 0;
			}
			seq[i] = count;
		}

		iBIT[tid] = SeqSize - 1 - seq[SeqSize - 1];
		BIT[tid][0] = seq[SeqSize - 1] / SeqSize;
		__syncthreads();

		// Build tree
		int before = BIT[tid][0];
		for (int i=1, offset=1; i<LOGT; i++, offset <<= 1){
			int tmp = tid - offset;
			if (tmp >= 0){
				if (before != 0 && BIT[tmp][i-1] != 0)
						before = (BIT[tid][i] = before + BIT[tmp][i-1]);
				else
						before = (BIT[tid][i] = 0);
			}
			else
				BIT[tid][i] = before;
			__syncthreads();
		}

		// Set
		int offset = tid - 1;
		for (int i=LOGT-1; i>=0 && offset>=0; i--)
			offset -= BIT[offset][i];

		if (offset >= 0) offset = (left + offset) * SeqSize + iBIT[offset];

		if (tid >= ThreadSize/2 || id < ThreadSize / 2){
			for (int i=0; i<SeqSize && seqIndex+i<n; i++){
				if (seq[i] == 0)
					offset = seqIndex + i;
				pos[seqIndex + i] = seqIndex + i - offset;
			}
		}
	}
}
 
void labeling(const char *text, int *pos, int text_size)
{
	dim3 grid(CeilDiv( CeilDiv(text_size, SeqSize), ThreadSize), 2), block(ThreadSize, 1);
	myCudaCount<<< grid, block>>>(text, pos, text_size);
}
