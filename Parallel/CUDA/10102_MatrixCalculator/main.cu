
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#define MAXN 1024
#define SeqSize 4

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }

__global__ void myMatrixMul(int N, uint32_t *cuC, uint32_t *cuA, uint32_t *cuTransB){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= N*N) return;
    int x = tid / N;
    int y = tid % N;
 
    uint32_t *_A = cuA + x*N;
    uint32_t *_B = cuTransB + y*N;
    uint32_t sum = 0;
    for (int k=N; k>0; k--)
        sum += *_A * *_B, _A++, _B++;
    cuC[tid] = sum;
}

__global__ void myMatrixAdd(int N, uint32_t *cuA, uint32_t *cuB){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
	cuA[tid] += cuB[tid];
}

void rand_gen(uint32_t c, int N, uint32_t *A) {
    uint32_t x = 2, n = N*N;
	uint32_t *_A = A;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++)
            _A[j] = x = (x * x + c + i + j)%n;
		_A += N;
    }
}

void rand_gen_t(uint32_t c, int N, uint32_t *A) {
    uint32_t x = 2, n = N*N;

    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
            A[j*N+i] = x = (x * x + c + i + j)%n;
}

uint32_t signature(int N, uint32_t *A) {
    uint32_t h = 0;
	uint32_t *_A = A;
	for (int i=N*N; i>0; i--, _A++)
		h = (h + *_A) * 2654435761LU;
    return h;
}

uint32_t IN[6][MAXN*MAXN];

int main() {
    int N, S[6];
    scanf("%d", &N);
    for (int i = 0; i < 6; i++)
		scanf("%d", &S[i]);

	#pragma omp parallel for
	for (int i=0; i<6; i++) {
		if (i == 0 || i == 2)
			rand_gen(S[i], N, IN[i]);
		else
			rand_gen_t(S[i], N, IN[i]);
	}

	uint32_t *cuIn[6], *cuTmp[4];
	for (int i=0; i<6; i++){
		hipMalloc( &cuIn[i], sizeof(uint32_t)*N*N );
		hipMemcpy( cuIn[i], IN[i], sizeof(uint32_t)*N*N, hipMemcpyHostToDevice);
	}
	for (int i=0; i<4; i++)
		hipMalloc( &cuTmp[i], sizeof(uint32_t)*N*N );

	dim3 grid(CeilDiv(N*N, 32)), block(32);
	// AB
	myMatrixMul<<< grid, block >>>(N, cuTmp[0], cuIn[0], cuIn[1]);
	// CD
	myMatrixMul<<< grid, block >>>(N, cuTmp[1], cuIn[2], cuIn[3]);
	// ABE
	myMatrixMul<<< grid, block >>>(N, cuTmp[3], cuTmp[0], cuIn[4]);
	// CDF
	myMatrixMul<<< grid, block >>>(N, cuTmp[4], cuTmp[1], cuIn[5]);
	// AB+CD
	myMatrixAdd<<< grid, block >>>(N, cuTmp[0], cuTmp[1]);
	// ABE+CDF
	myMatrixAdd<<< grid, block >>>(N, cuTmp[3], cuTmp[4]);

	hipMemcpy( IN[0], cuTmp[0], sizeof(uint32_t)*N*N, hipMemcpyDeviceToHost);
	hipMemcpy( IN[1], cuTmp[3], sizeof(uint32_t)*N*N, hipMemcpyDeviceToHost);

    printf("%u\n", signature(N, IN[0]));
    printf("%u\n", signature(N, IN[1]));

	for (int i=0; i<6; i++)
		hipFree(cuIn[i]);
	for (int i=0; i<4; i++)
		hipFree(cuTmp[i]);
    return 0;
}
