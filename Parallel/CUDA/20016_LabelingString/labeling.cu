#include "hip/hip_runtime.h"
#include "labeling.h"
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

#define ThreadSize 256
#define LOGT 4
#define SeqSize 4

__global__ void myCudaCount(const char *text, int *pos, int n){
	int left = blockIdx.x * blockDim.x;
	int tid = threadIdx.x;
	int id = left + tid;

	__shared__ int BIT[ThreadSize][LOGT+1];
	__shared__ int iBIT[ThreadSize];
	int seqIndex = id * SeqSize;
	int seq[SeqSize];

	if (seqIndex < n){
		// Transform
		for (int i=0, count=1; i<SeqSize; i++, count++){
			if (seqIndex + i < n){
				if (text[seqIndex + i] <= ' ')
					count = 0;
			}
			else
				count = 0;
			seq[i] = count;
		}

		iBIT[tid] = SeqSize - 1 - seq[SeqSize - 1];
		BIT[tid][0] = seq[SeqSize - 1];
		__syncthreads();

		// Build tree
		for (int i=1, offset=1; i<=LOGT; i++, offset<<=2){
			int count = 0;
			for (int j=0, tmp = tid; j<SeqSize && tmp >= 0; j++, tmp-=offset){
				if (BIT[tmp][i-1] >= SeqSize) count++;
				else break;
			}
			BIT[tid][i] = count;
			__syncthreads();
		}

		// Set
		int offset = tid - 1;
		for (int i=LOGT, shift=64; i>0 && offset>=0; i--, shift>>=2)
			offset -= shift * BIT[offset][i];

		if (offset >= 0) offset = (left + offset) * SeqSize + iBIT[offset];
		else offset = left * SeqSize - 1;

		for (int i=0; i<SeqSize && seqIndex+i<n; i++){
			if (seq[i] == 0) offset = seqIndex + i;
			pos[seqIndex + i] = seqIndex + i - offset;
		}
	}
}

__global__ void myCudaFix(int *pos, int n){
	int left = (blockIdx.x+1) * ThreadSize * SeqSize;
	int tid = threadIdx.x * SeqSize;
	int seqIndex = left + tid;
	if (seqIndex < n){
		int prefix = pos[left - 1];
		if (prefix == 0) return;
		if (pos[seqIndex] > tid) pos[seqIndex] += prefix, seqIndex++, tid++;
		if (pos[seqIndex] > tid) pos[seqIndex] += prefix, seqIndex++, tid++;
		if (pos[seqIndex] > tid) pos[seqIndex] += prefix, seqIndex++, tid++;
		if (pos[seqIndex] > tid) pos[seqIndex] += prefix;
	}
}

void labeling(const char *text, int *pos, int text_size)
{
	uint32_t nGrid = CeilDiv( CeilDiv(text_size, SeqSize), ThreadSize);
	dim3 grid(nGrid, 1), block(ThreadSize, 1);
	myCudaCount<<< grid, block>>>(text, pos, text_size);

	dim3 grid2( nGrid-1, 1), block2(ThreadSize/2, 1);
	myCudaFix<<< grid2, block2>>>(pos, text_size);
}
