#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <stdint.h>
#include "labeling.h"
 
//#define DEBUG
 
namespace {
    uint32_t seed = 0;
    void p_srand(uint32_t x) { seed = x;}
    uint32_t p_rand() {return seed = (seed*9301 + 49297);}
}
 
int main() {
    static const int MAXN = 16777216<<1;
    static const int MAXTOKEN = 500;
    static char s[MAXN];
    static int ret[MAXN];
    static char *cuStr;
    static int32_t *cuPos;
    int cases = 0;
    int n, m1, m2, seed;
    while (scanf("%d %d %d %d", &n, &m1, &m2, &seed) == 4) {
        assert(n <= MAXN);
        assert(m1 <= MAXTOKEN);
        p_srand(seed);
        hipMalloc(&cuStr, sizeof(char)*n);
        hipMalloc(&cuPos, sizeof(int32_t)*n);
        // random string
        {
            int pos = 0;
            for (; pos < n;) {
                int sp = pos == 0 ? p_rand()%m2 : p_rand()%m2+1;
                int cp = p_rand()%m1+1;
                for (; sp && pos < n; sp--, pos++)
                    s[pos] = ' ';
                for (; cp && pos < n; cp--, pos++)
                    s[pos] = 'a' + p_rand()%26;
            }
#ifdef DEBUG
            for (int i = 0; i < n; i++)
                putchar(s[i]);
            puts("");
#endif
            hipMemcpy(cuStr, s, sizeof(char)*n, hipMemcpyHostToDevice);
        }
        // test performance
        {
            clock_t st, ed;
            st = clock();
            const int ROUND = 3000;
            for (int i = 0; i < ROUND; i++)    {
                labeling(cuStr, cuPos, n);
            }
            hipMemcpy(ret, cuPos, sizeof(int32_t)*n, hipMemcpyDeviceToHost);
            ed = clock() - st;
            fprintf(stderr, "It took average %lf seconds.\n", ((float) ed)/CLOCKS_PER_SEC/ROUND);
        }
        // check
        {
            clock_t st, ed;
            st = clock();
            uint32_t HEX = 0;
            for (int i = 0, sum = 0; i < n; i++) {
#ifdef DEBUG
                // printf("%d%c", ret[i], " \n"[i==n-1]);
                printf("%d", ret[i]);
                if (i == n-1)
                    puts("");
#endif
                if (s[i] > ' ')
                    sum++;
                else
                    sum = 0;
//printf("i = %d, sum = %d, ret = %d\n", i, sum, ret[i]);
                assert(sum == ret[i]);
                HEX ^= i*ret[i];
            }
            ed = clock() - st;
            fprintf(stderr, "Check task took %lf seconds.\n", ((float) ed)/CLOCKS_PER_SEC);
            printf("Case #%d: PASS %X\n", ++cases, HEX);
        }
        hipFree(cuPos);
        hipFree(cuStr);
    }
 
    return 0;
}
