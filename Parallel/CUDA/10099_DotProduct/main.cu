#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <inttypes.h>
#include <stdint.h>
#include <thrust/execution_policy.h>
#include <thrust/reduce.h>

#define MAXN 16777216
#define ThreadSize 256
#define SeqSize 1024
#define atomicN ThreadSize

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ uint32_t rotate_left(uint32_t x, uint32_t n) {
    return  (x << n) | (x >> (32-n));
}
__device__ __host__ uint32_t encrypt(uint32_t m, uint32_t key) {
    return (rotate_left(m, key&31) + key)^key;
}

#define DOT { sum += encrypt(i, key1) * encrypt(i, key2), i++; }
#define UNROLL2  {DOT DOT}
#define UNROLL4  {UNROLL2 UNROLL2}
#define UNROLL8  {UNROLL4 UNROLL4}
#define UNROLL16 {UNROLL8 UNROLL8}

__global__ void myDotProduct( uint32_t *cuC, int n, uint32_t key1, uint32_t key2){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int id = tid * SeqSize;
	int bound = (id + SeqSize < n)? id + SeqSize: n;

	uint32_t sum = 0;
	int i;
	for (i=id; i+15<bound; ){
		UNROLL16;
	}
	if (i+7 < bound) UNROLL8;
	if (i+3 < bound) UNROLL4;
	if (i+1 < bound) UNROLL2;
	while (i < bound) DOT;

	atomicAdd( &(cuC[tid % atomicN]), sum);
}

int main(int argc, char *argv[]) {
    int N;
    uint32_t key1, key2;
	uint32_t *cuC;
	hipMalloc( &cuC, sizeof(uint32_t)*atomicN);
    while (scanf("%d %" PRIu32 " %" PRIu32, &N, &key1, &key2) == 3) {
		int chunk = CeilDiv(N, SeqSize);
		dim3 grid(CeilDiv(chunk, ThreadSize)), block(ThreadSize);

		hipMemset( cuC, 0, sizeof(uint32_t)*atomicN);
		myDotProduct<<<grid, block>>>(cuC, N, key1, key2);
		uint32_t sum = thrust::reduce(thrust::device, cuC, cuC+atomicN);
        printf("%" PRIu32 "\n", sum);
    }
	hipFree(cuC);
    return 0;
}
