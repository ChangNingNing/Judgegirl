
#include <hip/hip_runtime.h>
#include <stdio.h>

#define MAXN 2003

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }

__device__ int neighbor(int index, int n, char* cuT){
	int num;
	num = cuT[index-n-1] + cuT[index-n-0] + cuT[index-n+1]
		+ cuT[index-0-1] 				  + cuT[index-0+1]
		+ cuT[index+n-1] + cuT[index+n-0] + cuT[index+n+1];
	return num;
}

__global__ void myGOL( char *cuT, int W, int N){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= N*N) return;

	int n = N + 2;
	int x = tid / N + 1, y = tid % N + 1;
	int R = W ^ 1;
	int Rindex = x*n+y + R*n*n;
	int Windex = x*n+y + W*n*n;

	int num = neighbor( Rindex, n, cuT );
	cuT[Windex] = cuT[Rindex]? num==2 || num==3: num==3;
}

int main(){
	int N, M;
	static char T[MAXN*MAXN] = {0};
	char *cuT;

	scanf("%d %d", &N, &M);
	int n = N + 2;
	for (int i=1; i<=N; i++)
		scanf("%s", T+i*n+1);
	for (int i=0; i<n; i++)
		for (int j=0; j<n; j++)
			T[i*n+j] = T[i*n+j]=='1'? 1: 0;

	hipMalloc( &cuT, sizeof(char)*n*n*2 );
	hipMemset( cuT, 0, sizeof(char)*n*n*2 );
	hipMemcpy( cuT, T, sizeof(char)*n*n, hipMemcpyHostToDevice );

	int flag = 0;
	dim3 grid(CeilDiv(N*N, 256)), block(256);
	for (int i=0; i<M; i++){
		flag ^= 1;
		myGOL<<< grid, block >>>( cuT, flag, N );
	}

	hipMemcpy( T, cuT + flag*n*n, sizeof(char)*n*n, hipMemcpyDeviceToHost );
	hipFree(cuT);

	static char out[MAXN*MAXN];
	int length = 0;
	for (int i=1; i<=N; i++){
		for (int j=1; j<=N; j++)
			out[length++] = T[i*n+j] + '0';
		out[length++] = '\n';
	}
	out[--length] = '\0';
	puts(out);
	return 0;
}
