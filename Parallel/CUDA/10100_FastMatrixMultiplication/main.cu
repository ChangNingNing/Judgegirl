
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define MAXN 1024

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }

void rand_gen(uint32_t cA, uint32_t cB, int N, uint32_t *A, uint32_t *B) {
    uint32_t xA = 2, n = N*N;
	uint32_t xB = 2;

	uint32_t *_A = A;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            *_A = xA = (xA * xA + cA + i + j)%n, _A++;
			B[j*N + i] = xB = (xB * xB + cB + i + j)%n;
        }
    }
}

uint32_t signature(int N, uint32_t *A) {
    uint32_t h = 0;
	uint32_t *_A = A;
    for (int i = N*N; i > 0; i--)
		h = (h + *_A) * 2654435761LU, _A++;
    return h;
}

__global__ void myMatrixMul(int N, uint32_t *cuC, uint32_t *cuA, uint32_t *cuTransB){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= N*N) return;
	int x = tid / N;
	int y = tid % N;

	uint32_t *_A = cuA + x*N;
	uint32_t *_B = cuTransB + y*N;
	uint32_t sum = 0;
	for (int k=N; k>0; k--)
		sum += *_A * *_B, _A++, _B++;
	cuC[x*N + y] = sum;
}

uint32_t A[MAXN*MAXN], B[MAXN*MAXN], C[MAXN*MAXN];

int main() {
    int N;
    uint32_t S1, S2;
    scanf("%d %u %u", &N, &S1, &S2);
    rand_gen(S1, S2, N, A, B);

	uint32_t *cuA, *cuB, *cuC;
	hipMalloc( &cuA, sizeof(uint32_t)*N*N );
	hipMalloc( &cuB, sizeof(uint32_t)*N*N );
	hipMalloc( &cuC, sizeof(uint32_t)*N*N );

	hipMemcpy( cuA, A, sizeof(uint32_t)*N*N, hipMemcpyHostToDevice);
	hipMemcpy( cuB, B, sizeof(uint32_t)*N*N, hipMemcpyHostToDevice);

	dim3 grid(CeilDiv(N*N, 32)), block(32);
	myMatrixMul<<< grid, block >>>( N, cuC, cuA, cuB );
	
	hipMemcpy( C, cuC, sizeof(uint32_t)*N*N, hipMemcpyDeviceToHost);
    printf("%u\n", signature(N, C));

	hipFree(cuA);
	hipFree(cuB);
	hipFree(cuC);
    return 0;
}
